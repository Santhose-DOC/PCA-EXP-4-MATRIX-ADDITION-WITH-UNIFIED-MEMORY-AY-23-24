#include "hip/hip_runtime.h"
%%cu
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

#ifndef _COMMON_H
#define _COMMON_H

#define CHECK(call)                                                            \
{                                                                              \
    const hipError_t error = call;                                            \
    if (error != hipSuccess)                                                  \
    {                                                                          \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
        fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                hipGetErrorString(error));                                    \
        exit(1);                                                               \
    }                                                                          \
}

inline double seconds()
{
    struct timeval tp;
    struct timezone tzp;
    int i = gettimeofday(&tp, &tzp);
    return ((double)tp.tv_sec + (double)tp.tv_usec * 1.e-6);
}

#endif // _COMMON_H

// Initialize matrix with random float values
void initialData(float *ip, const int size)
{
    for (int i = 0; i < size; i++)
    {
        ip[i] = (float)(rand() & 0xFF) / 10.0f;
    }
}

// Matrix addition on CPU
void sumMatrixOnHost(float *A, float *B, float *C, const int nx, const int ny)
{
    float *ia = A;
    float *ib = B;
    float *ic = C;

    for (int iy = 0; iy < ny; iy++)
    {
        for (int ix = 0; ix < nx; ix++)
        {
            ic[ix] = ia[ix] + ib[ix];
        }

        ia += nx;
        ib += nx;
        ic += nx;
    }
}

// Result check between host and GPU results
void checkResult(float *hostRef, float *gpuRef, const int N)
{
    double epsilon = 1.0E-8;
    bool match = true;

    for (int i = 0; i < N; i++)
    {
        if (abs(hostRef[i] - gpuRef[i]) > epsilon)
        {
            match = false;
            printf("Mismatch at index %d: host %f vs gpu %f\n", i, hostRef[i], gpuRef[i]);
            break;
        }
    }

    if (!match)
    {
        printf("Arrays do not match.\n\n");
    }
    else
    {
        printf("Arrays match.\n\n");
    }
}

// Matrix addition kernel on GPU
__global__ void sumMatrixGPU(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    // Calculate thread's absolute index in 2D grid
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int idx = iy * nx + ix;

    // Check if we are within the matrix bounds
    if (ix < nx && iy < ny)
    {
        MatC[idx] = MatA[idx] + MatB[idx];
    }
}

int main(int argc, char **argv)
{
    printf("%s Starting...\n", argv[0]);

    // Set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    // Set up matrix size
    int nx, ny;
    int ishift = 12;  // Default: 2^12 = 4096

    if (argc > 1) ishift = atoi(argv[1]);

    nx = ny = 1 << ishift;  // square matrix
    int nxy = nx * ny;
    int nBytes = nxy * sizeof(float);
    printf("Matrix size: nx = %d, ny = %d\n", nx, ny);

    // Allocate Unified Memory
    float *A, *B, *hostRef, *gpuRef;
    CHECK(hipMallocManaged((void **)&A, nBytes));
    CHECK(hipMallocManaged((void **)&B, nBytes));
    CHECK(hipMallocManaged((void **)&hostRef, nBytes));
    CHECK(hipMallocManaged((void **)&gpuRef,  nBytes));

    // Initialize data
    double iStart = seconds();
    initialData(A, nxy);
    initialData(B, nxy);
    double iElaps = seconds() - iStart;
    printf("Initialization:\t\t%f sec\n", iElaps);

    memset(hostRef, 0, nBytes);
    memset(gpuRef, 0, nBytes);

    // Host computation
    iStart = seconds();
    sumMatrixOnHost(A, B, hostRef, nx, ny);
    iElaps = seconds() - iStart;
    printf("sumMatrix on host:\t%f sec\n", iElaps);

    // Kernel launch config
    int dimx = 32;
    int dimy = 32;
    dim3 block(dimx, dimy);
    dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);

    // Warm-up kernel
    sumMatrixGPU<<<grid, block>>>(A, B, gpuRef, 1, 1);
    CHECK(hipDeviceSynchronize());

    // Actual kernel launch
    iStart = seconds();
    sumMatrixGPU<<<grid, block>>>(A, B, gpuRef, nx, ny);
    CHECK(hipDeviceSynchronize());
    iElaps = seconds() - iStart;
    printf("sumMatrix on GPU:\t%f sec <<<(%d,%d), (%d,%d)>>>\n",
           iElaps, grid.x, grid.y, block.x, block.y);

    // Check for kernel errors
    CHECK(hipGetLastError());

    // Compare results
    checkResult(hostRef, gpuRef, nxy);

    // Cleanup
    CHECK(hipFree(A));
    CHECK(hipFree(B));
    CHECK(hipFree(hostRef));
    CHECK(hipFree(gpuRef));

    CHECK(hipDeviceReset());
    return 0;
}